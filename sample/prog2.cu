#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>

__global__ void
vector(int *A, int *B, int *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[B[i]] = A[i];
    }
}

int main(int argc, char **argv)
{
    struct timeval start, end;
    hipError_t err = hipSuccess;

    int numElements = 5000000;
    size_t size = numElements * sizeof(int);

    int *h_A = (int *)malloc(size);
    int *h_B = (int *)malloc(size);
    int *h_C = (int *)malloc(size);
    if (h_A == NULL || h_B == NULL || h_C == NULL) fprintf(stderr, "Failed to allocate host vectors!\n");

    // Initialize the host input vectors
    srand(time(NULL));
    for (int i = 0; i < numElements; i++)
    {
        h_A[i] = i;
        h_B[i] = rand()%numElements;
    }

    // Allocate the device input vector A
    int *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess) fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));

    // Allocate the device input vector B
    int *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);
    if (err != hipSuccess) fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));

    // Allocate the device output vector C
    int *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);
    if (err != hipSuccess) fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));

    // Copy the host input vectors A and B in host memory to the device input vectors in device memory
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 512;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
 //   printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    
    gettimeofday(&start, NULL);
    vector<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    gettimeofday(&end, NULL);
    printf("%ld\n", ((end.tv_sec * 1000000 + end.tv_usec)
		  - (start.tv_sec * 1000000 + start.tv_usec)));
    err = hipGetLastError();
    if (err != hipSuccess) fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));

    // Copy the device result vector in device memory to the host result vector in host memory.
 //   printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));

    // Verify that the result vector is correct
   for (int i = 0; i < numElements; i++) fprintf(stderr, "Element in C[%d] is %d!\n", i, h_C[i]);

    // Free device global memory
    err = hipFree(d_A);
    if (err != hipSuccess) fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
    err = hipFree(d_B);
    if (err != hipSuccess) fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
    err = hipFree(d_C);
    if (err != hipSuccess) fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    err = hipDeviceReset();
    if (err != hipSuccess) fprintf(stderr, "Failed to deinitialize the device! error = %s\n", hipGetErrorString(err));
}

